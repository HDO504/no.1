#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>

#define MAX 1024
#define M 2

static __device__ __inline__ unsigned int __mysmid(){
	unsigned int smid;
	asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
	return smid;

}

__global__ void testKernel(int SM_num_start, int SM_num_end,int val)
{
    int SM_num;
    SM_num = __mysmid();

    if((SM_num_start <= SM_num)&&(SM_num <= SM_num_end)){


    
	

    long int off_set = blockDim.x * (SM_num_end - SM_num_start + 1);
    
    int id = threadIdx.x + (SM_num - SM_num_end)* blockDim.x;


    for(id = id;id < MAX; id = id + off_set){
    printf("[%d, %d]:\t\tValue is:%d\n",\
            blockIdx.x,\
            threadIdx.x,\
            val);
    }
  }
}

int main(int argc, char **argv)
{
 /*
    int devID;
    cudaDeviceProp props;

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    //Get GPU information
    checkCudaErrors(cudaGetDevice(&devID));
    checkCudaErrors(cudaGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
           devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");

    //Kernel configuration, where a two-dimensional grid and
    //three-dimensional blocks are configured.
  */

    int SM_num_start = 0;
    int SM_num_end = 2;
    testKernel<<<1024/32, 32>>>(SM_num_start,SM_num_end ,10);
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}


