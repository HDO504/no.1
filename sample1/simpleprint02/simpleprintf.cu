#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>

#define MAX 1024
#define M 2

__global__ void testKernel(int val)
{
    long int off_set = blockDim.x * M;
    
    int id = blockIdx.x * blockDim.x *M;


    for(id = id;id < MAX; id = id + off_set){
    printf("[%d, %d]:\t\tValue is:%d\n",\
            blockIdx.x,\
            threadIdx.x,\
            val);
    }
}

int main(int argc, char **argv)
{
 /*
    int devID;
    cudaDeviceProp props;

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    //Get GPU information
    checkCudaErrors(cudaGetDevice(&devID));
    checkCudaErrors(cudaGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
           devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");

    //Kernel configuration, where a two-dimensional grid and
    //three-dimensional blocks are configured.
  */
    testKernel<<<M, 32>>>(10);
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}


